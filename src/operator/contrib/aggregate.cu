#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */
 /*!
 * Copyright (c) 2018 by Contributors
 * \file encoding.cc
 * \brief Encoding Layer
 * \author Hang Zhang
 */

namespace mxnet {
namespace op {

static const unsigned WARP_SIZE = 32;

// The maximum number of threads in a block
static const unsigned MAX_BLOCK_SIZE = 512U;

template<typename In, typename Out>
struct ScalarConvert {
  static __host__ __device__ __forceinline__ Out to(const In v) { return (Out) v; }
};

// Number of threads in a block given an input size up to MAX_BLOCK_SIZE
static unsigned getNumThreads(int nElem, const bool smaller) {
  unsigned threadSizes[5] = {32, 64, 128, 256, MAX_BLOCK_SIZE};
  const int maxi = smaller ? 4 : 5;
  for (int i = 0; i != maxi; ++i) {
    if (static_cast<unsigned>(nElem) <= threadSizes[i]) {
      return threadSizes[i];
    }
  }
  return smaller ? (MAX_BLOCK_SIZE >> 1) : MAX_BLOCK_SIZE;
}

// Returns the index of the most significant 1 bit in `val`.
__device__ __forceinline__ int getMSB(int val) {
  return 31 - __clz(val);
}

#if CUDA_VERSION >= 9000
#define FULLMASK 0xFFFFFFFF
#define __shfl_xor(...) __shfl_xor_sync(FULLMASK, __VA_ARGS__)
#endif

// Sum across all threads within a warp
template<typename T>
static __device__ __forceinline__ T warpSum(T val) {
#if __CUDA_ARCH__ >= 300
for (int i = 0; i < getMSB(WARP_SIZE); ++i) {
    val += __shfl_xor(val, 1 << i, WARP_SIZE);
  }
#else
__shared__ T values[MAX_BLOCK_SIZE];
values[threadIdx.x] = val;
__threadfence_block();
const int base = (threadIdx.x / WARP_SIZE) * WARP_SIZE;
for (int i = 1; i < WARP_SIZE; i++) {
val += values[base + ((i + threadIdx.x) % WARP_SIZE)];
}
#endif
return val;
}

template<typename DType, typename AccReal>
struct Float2 {
  AccReal v1, v2;
  __device__ Float2() {}
  __device__ Float2(DType v1, DType v2)
    : v1(ScalarConvert<DType, AccReal>::to(v1))
      , v2(ScalarConvert<DType, AccReal>::to(v2)) {}
  __device__ Float2(DType v)
    : v1(ScalarConvert<DType, AccReal>::to(v))
      , v2(ScalarConvert<DType, AccReal>::to(v)) {}
  __device__ Float2(int v)
    : v1(ScalarConvert<int, AccReal>::to(v))
      , v2(ScalarConvert<int, AccReal>::to(v)) {}
  __device__ Float2 &operator+=(const Float2 &a) {
    v1 += a.v1;
    v2 += a.v2;
    return *this;
  }
};

template<typename DType, typename AccReal>
static __device__ __forceinline__ Float2<DType, AccReal> warpSum(Float2<DType, AccReal> value) {
  value.v1 = warpSum(value.v1);
  value.v2 = warpSum(value.v2);
  return value;
}

// Sum across (batch, x/y/z) applying Op() pointwise
template<typename T, typename Op>
static __device__ T reduce(Op op, DeviceTensor tensor, int plane) {
  T sum = (T) 0;
  for (int batch = 0; batch < tensor.OuterSize(); ++batch) {
    for (int x = threadIdx.x; x < tensor.InnerSize(); x += blockDim.x) {
      sum += op(batch, plane, x);
    }
  }

  // sum over NumThreads within a warp
  sum = warpSum(sum);

  // 'transpose', and reduce within warp again
  __shared__ T shared[32];
  __syncthreads();
  if (threadIdx.x % WARP_SIZE == 0) {
    shared[threadIdx.x / WARP_SIZE] = sum;
  }
  if (threadIdx.x >= blockDim.x / WARP_SIZE && threadIdx.x < WARP_SIZE) {
    // zero out the other entries in shared
    shared[threadIdx.x] = (T) 0;
  }
  __syncthreads();
  if (threadIdx.x / WARP_SIZE == 0) {
    sum = warpSum(shared[threadIdx.x]);
    if (threadIdx.x == 0) {
      shared[0] = sum;
    }
  }
  __syncthreads();

  // Everyone picks it up, should be broadcast into the whole gradInput
  return shared[0];
}

template<typename xpu, typename real, typename AccReal>
__global__ void Aggregate_Forward_kernel (
    DeviceTensor<real, 3> E,
    DeviceTensor<real, 3> A,
    DeviceTensor<real, 3> X,
    DeviceTensor<real, 2> C) {
    /* declarations of the variables */
    int b, k, d, N;
    /* Get the index and channels */ 
    b = blockIdx.z;
    d = blockIdx.x;
    k = blockIdx.y;
    N = X.getSize(1);

    /* main operation */
    Encoding_(AggOp) g(A,X,C);
    E[b][k][d] = Encoding_(reduce_agg)(g,b,k,d,N);
}


template<typename xpu, typename real, typename AccReal>
__global__ void Aggregate_Backward_kernel (
    DeviceTensor<real, 3> GA,
    DeviceTensor<real, 3> GE,
    DeviceTensor<real, 3> A,
    DeviceTensor<real, 3> X,
    DeviceTensor<real, 2> C) {
    /* declarations of the variables */
    int b, k, i, D;
    /* Get the index and channels */ 
    b = blockIdx.z;
    i = blockIdx.y;
    k = blockIdx.x;
    D = GE.getSize(2);
    /* main operation */
    Encoding_(AggBackOp) g(GE,X,C);
    GA[b][i][k] = Encoding_(reduce_aggback)(g,b,i,k,D);
}

}  // namespace op
}  // namespace mxnet
